#include "include/linear.h"
#include "include/common.h"
#include "include/A8W4_gemm.h"
#include "torch_int/kernels/include/cutlass_extensions/include/cutlass_extensions/epilogue/threadblock/epilogue_per_row_per_col_scale.h"
#include "torch_int/kernels/include/cutlass_extensions/include/cutlass_extensions/gemm/kernel/gemm_with_epilogue_visitor.h"
#include "torch_int/kernels/include/cutlass_extensions/include/cutlass_extensions/epilogue_helpers.h"

#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/half.h>

#include <cutlass/gemm/device/default_gemm_configuration.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/gemm/device/gemm_universal_base.h>
#include <cutlass/gemm/kernel/default_gemm.h>
#include <cutlass/epilogue/threadblock/epilogue_with_visitor.h>
#include <cutlass/epilogue/thread/linear_combination_relu.h>
#include <cutlass/numeric_types.h>
#include <cutlass/util/host_tensor.h>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>


#define THREAD_PER_BLOCK 1024

inline int GET_BLOCKS(const int N) {
  int optimal_block_num = (N + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;

  return optimal_block_num;
}



// used by out_proj and fc2, return FP32
torch::Tensor linear_a8_w8_bbf16_obf16(torch::Tensor input,  // INT8
                                       torch::Tensor weight, // INT8
                                       torch::Tensor bias,   // BF16
                                       float alpha   //
) {
  auto M = input.size(0);
  auto N = weight.size(0);
  auto K = input.size(1);

  using ElementOutput = cutlass::bfloat16_t;
  using ElementAccumulator = int32_t;
  using ElementComputeEpilogue = cutlass::bfloat16_t;
  using ElementInputA = int8_t; // <- data type of elements in input matrix A
  using ElementInputB = int8_t; // <- data type of elements in input matrix B

  // The code section below describes matrix layout of input and output
  // matrices. Column Major for Matrix A, Row Major for Matrix B and Row Major
  // for Matrix C
  using LayoutInputA = cutlass::layout::RowMajor;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  using LayoutOutput = cutlass::layout::RowMajor;

  using Gemm = cutlass::gemm::device::Gemm<
      int8_t, cutlass::layout::RowMajor, int8_t, cutlass::layout::ColumnMajor,
      ElementOutput, cutlass::layout::RowMajor, ElementAccumulator,
      cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<256, 128, 64>,
      cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementComputeEpilogue,cutlass::epilogue::thread::ScaleType::NoBetaScaling>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3>;

  auto input_size = cutlass::MatrixCoord(M, K);
  auto weight_size = cutlass::MatrixCoord(K, N);
  auto output_size = cutlass::MatrixCoord(M, N);

  auto device = input.device();
  // use the broadcasted bias as the output
  auto out = bias.to(device).view({1, -1}).repeat({M, 1});

  // constexpr int kSparse = Gemm::kSparse;
  // How many elements of A are covered per ElementE
  // constexpr int kElementsPerElementE = Gemm::kElementsPerElementE;
  // The size of individual meta data
  // constexpr int kMetaSizeInBits = Gemm::kMetaSizeInBits;
  cutlass::gemm::GemmCoord problem_size(M, N, K);

  cutlass::TensorRef<ElementInputA, LayoutInputA> input_ref(
      input.data_ptr<ElementInputA>(), LayoutInputA::packed(input_size));
  cutlass::TensorRef<ElementInputB, LayoutInputB> weight_ref(
      weight.data_ptr<ElementInputB>(), LayoutInputB::packed(weight_size));
  cutlass::TensorRef<ElementOutput, LayoutOutput> out_ref(
      reinterpret_cast<ElementOutput* const> (out.data_ptr()), LayoutOutput::packed(output_size));
      
  ElementComputeEpilogue alpha_c = ElementComputeEpilogue(alpha);

  typename Gemm::Arguments arguments{
      problem_size, // <- problem size of matrix multiplication
      input_ref,    // <- reference to matrix A on device
      weight_ref,   // <- reference to matrix B on device
      out_ref,      // <- reference to matrix C on device
      out_ref,      // <- reference to matrix D on device
      {alpha_c}, 1};
  Gemm gemm_op;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  cutlass::Status status = gemm_op.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm_op.initialize(arguments, workspace.get());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm_op();
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot run");
  }

  return out;
}


// used by out_proj and fc2, return FP32
torch::Tensor linear_a8_w8_bfp16_ofp16(torch::Tensor input,  // INT8
                                       torch::Tensor weight, // INT8
                                       torch::Tensor bias,   // BF16
                                       float alpha   //
) {
  auto M = input.size(0);
  auto N = weight.size(0);
  auto K = input.size(1);

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = int32_t;
  using ElementComputeEpilogue = cutlass::half_t;
  using ElementInputA = int8_t; // <- data type of elements in input matrix A
  using ElementInputB = int8_t; // <- data type of elements in input matrix B

  // The code section below describes matrix layout of input and output
  // matrices. Column Major for Matrix A, Row Major for Matrix B and Row Major
  // for Matrix C
  using LayoutInputA = cutlass::layout::RowMajor;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  using LayoutOutput = cutlass::layout::RowMajor;

  using Gemm = cutlass::gemm::device::Gemm<
      int8_t, cutlass::layout::RowMajor, int8_t, cutlass::layout::ColumnMajor,
      ElementOutput, cutlass::layout::RowMajor, ElementAccumulator,
      cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<256, 128, 64>,
      cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementComputeEpilogue,cutlass::epilogue::thread::ScaleType::NoBetaScaling>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3>;

  auto input_size = cutlass::MatrixCoord(M, K);
  auto weight_size = cutlass::MatrixCoord(K, N);
  auto output_size = cutlass::MatrixCoord(M, N);

  auto device = input.device();
  // use the broadcasted bias as the output
  auto out = bias.to(device).view({1, -1}).repeat({M, 1});

  // constexpr int kSparse = Gemm::kSparse;
  // How many elements of A are covered per ElementE
  // constexpr int kElementsPerElementE = Gemm::kElementsPerElementE;
  // The size of individual meta data
  // constexpr int kMetaSizeInBits = Gemm::kMetaSizeInBits;
  cutlass::gemm::GemmCoord problem_size(M, N, K);

  cutlass::TensorRef<ElementInputA, LayoutInputA> input_ref(
      input.data_ptr<ElementInputA>(), LayoutInputA::packed(input_size));
  cutlass::TensorRef<ElementInputB, LayoutInputB> weight_ref(
      weight.data_ptr<ElementInputB>(), LayoutInputB::packed(weight_size));
  cutlass::TensorRef<ElementOutput, LayoutOutput> out_ref(
      reinterpret_cast<ElementOutput* const> (out.data_ptr()), LayoutOutput::packed(output_size));

  ElementComputeEpilogue alpha_c = ElementComputeEpilogue(alpha); 

  typename Gemm::Arguments arguments{
      problem_size, // <- problem size of matrix multiplication
      input_ref,    // <- reference to matrix A on device
      weight_ref,   // <- reference to matrix B on device
      out_ref,      // <- reference to matrix C on device
      out_ref,      // <- reference to matrix D on device
      {alpha_c}, 1};
  Gemm gemm_op;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  cutlass::Status status = gemm_op.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm_op.initialize(arguments, workspace.get());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm_op();
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot run");
  }

  return out;
}


// used by out_proj and fc2, return FP32
torch::Tensor linear_a8_w8_bfp32_ofp32(torch::Tensor input,  // INT8
                                       torch::Tensor weight, // INT8
                                       torch::Tensor bias,   // BF16
                                       float alpha   // 
) {
  auto M = input.size(0);
  auto N = weight.size(0);
  auto K = input.size(1);

  using ElementOutput = float;
  using ElementAccumulator = int32_t;
  using ElementComputeEpilogue = float;
  using ElementInputA = int8_t; // <- data type of elements in input matrix A
  using ElementInputB = int8_t; // <- data type of elements in input matrix B

  // The code section below describes matrix layout of input and output
  // matrices. Column Major for Matrix A, Row Major for Matrix B and Row Major
  // for Matrix C
  using LayoutInputA = cutlass::layout::RowMajor;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  using LayoutOutput = cutlass::layout::RowMajor;

  using Gemm = cutlass::gemm::device::Gemm<
      int8_t, cutlass::layout::RowMajor, int8_t, cutlass::layout::ColumnMajor,
      ElementOutput, cutlass::layout::RowMajor, ElementAccumulator,
      cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<256, 128, 64>,
      cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator,ElementComputeEpilogue,cutlass::epilogue::thread::ScaleType::NoBetaScaling>, //cutlass::epilogue::thread::ScaleType::NoBetaScaling
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3>;

  auto input_size = cutlass::MatrixCoord(M, K);
  auto weight_size = cutlass::MatrixCoord(K, N);
  auto output_size = cutlass::MatrixCoord(M, N);

  auto device = input.device();
  // use the broadcasted bias as the output
  auto out = bias.to(device).view({1, -1}).repeat({M, 1});

  // constexpr int kSparse = Gemm::kSparse;
  // How many elements of A are covered per ElementE
  // constexpr int kElementsPerElementE = Gemm::kElementsPerElementE;
  // The size of individual meta data
  // constexpr int kMetaSizeInBits = Gemm::kMetaSizeInBits;
  cutlass::gemm::GemmCoord problem_size(M, N, K);

  cutlass::TensorRef<ElementInputA, LayoutInputA> input_ref(
      input.data_ptr<ElementInputA>(), LayoutInputA::packed(input_size));
  cutlass::TensorRef<ElementInputB, LayoutInputB> weight_ref(
      weight.data_ptr<ElementInputB>(), LayoutInputB::packed(weight_size));
  cutlass::TensorRef<ElementOutput, LayoutOutput> out_ref(
      reinterpret_cast<ElementOutput* const> (out.data_ptr()), LayoutOutput::packed(output_size));

  ElementComputeEpilogue alpha_c = ElementComputeEpilogue(alpha);
  
  typename Gemm::Arguments arguments{
      problem_size, // <- problem size of matrix multiplication
      input_ref,    // <- reference to matrix A on device
      weight_ref,   // <- reference to matrix B on device
      out_ref,      // <- reference to matrix C on device
      out_ref,      // <- reference to matrix D on device
      {alpha_c}, 1}; //reinterpret_cast<ElementComputeEpilogue* const> (alpha.data_ptr())
  Gemm gemm_op;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  cutlass::Status status = gemm_op.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm_op.initialize(arguments, workspace.get());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm_op();
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot run");
  }

  return out;
}
